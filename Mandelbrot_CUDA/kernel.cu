#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>

#include <stdio.h>
#include "SDL2-2.0.9/include/SDL.h"

#include <complex>

#include "Color.cpp"
#include "Vector2.cpp"

#include <thrust/complex.h>


using namespace thrust;

Vector2<int> windowSize(900, 600);
int iterations = 2000;

double minReal = -2;
double maxReal = 1;
double minImag = -1;
double maxImag = 1;

complex<double> zoomInPoint(-0.7336438924199521, 0.2455211406714035);

float zoomInFactor = 0.3;

void updateBoundaries() {
  minReal += zoomInFactor * (zoomInPoint.real() - minReal);
  maxReal -= zoomInFactor * (maxReal - zoomInPoint.real());
  minImag += zoomInFactor * (zoomInPoint.imag() - minImag);
  maxImag -= zoomInFactor * (maxImag - zoomInPoint.imag());

  printf("%f + %fi, %f + %fi\n", minReal, minImag, maxReal, maxImag);
}

__device__ complex<double> pixelToComplex(double minReal, double maxReal,
                                          double minImag, double maxImag, int x,
                                          int y, int pixelsX, int pixelsY) {
  double real = ((double)x / pixelsX) * (maxReal - minReal) + minReal;
  double imag = maxImag - ((double)y / pixelsY) * (maxImag - minImag);
  return complex<double>(real, imag);
}

__device__ unsigned char computeGrayscale(complex<double> c, int iterations) {
  complex<double> c2 = complex<double>(0, 0);
  for (int i = 0; i < iterations; i++) {
    c2 = c2 * c2 + c;
    if (abs(c2) > 4) {
      return 255 - 255 * i / iterations;
    }
  }
  return 0;
}

__global__ void computeGrayScaleValues(unsigned char* pixelValues, int* pixelsX,
                                       double* minReal, double* maxReal,
                                       double* minImag, double* maxImag,
                                       int* pixelsY, int* iterations) {
  for (int i = threadIdx.x; i < *pixelsX; i += blockDim.x) {
    for (int j = blockIdx.x; j < *pixelsY; j += gridDim.x) {
      unsigned char grayScale =
          computeGrayscale(pixelToComplex(*minReal, *maxReal, *minImag,
                                          *maxImag, i, j, *pixelsX, *pixelsY),
                           *iterations);

      pixelValues[j * *pixelsX + i] = grayScale;
    }
  }
}

void drawMandelbrot(SDL_Renderer* renderer, Vector2<int> size,
                    unsigned char* pixelValues) {
  for (int i = 0; i < size.x; i++) {
    for (int j = 0; j < size.y; j++) {
      int grayScale = pixelValues[j * size.x + i];
      // printf("%d %d : %d\n", i, j, grayScale);
      SDL_SetRenderDrawColor(renderer, grayScale, grayScale, grayScale, 255);
      SDL_RenderDrawPoint(renderer, i, j);
    }
  }
  SDL_RenderPresent(renderer);
}

int main(int argc, char** argv) {
  SDL_Window* window;
  SDL_Renderer* renderer;
  SDL_CreateWindowAndRenderer(windowSize.x, windowSize.y, 0, &window,
                              &renderer);

  int* devPixelsX;
  int* devPixelsY;
  double* devMinReal;
  double* devMaxReal;
  double* devMinImag;
  double* devMaxImag;
  int* devIterations;

  unsigned char* pixelValues = new unsigned char[windowSize.x * windowSize.y];
  unsigned char* devPixelValues =
      new unsigned char[windowSize.x * windowSize.y];

  hipMalloc((void**)&devPixelsX, sizeof(int));
  hipMalloc((void**)&devPixelsY, sizeof(int));
  hipMalloc((void**)&devMinReal, sizeof(double));
  hipMalloc((void**)&devMaxReal, sizeof(double));
  hipMalloc((void**)&devMinImag, sizeof(double));
  hipMalloc((void**)&devMaxImag, sizeof(double));
  hipMalloc((void**)&devIterations, sizeof(int));
  hipMalloc(&devPixelValues,
             windowSize.x * windowSize.y * sizeof(unsigned char));

  for (int i = 0; i < 12; i++) {
    
  }

  while (true) {
    hipMemcpy(devPixelsX, &windowSize.x, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devPixelsY, &windowSize.y, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devMinReal, &minReal, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devMaxReal, &maxReal, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devMinImag, &minImag, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devMaxImag, &maxImag, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devIterations, &iterations, sizeof(int), hipMemcpyHostToDevice);

    computeGrayScaleValues<<<512, 128>>>(devPixelValues, devPixelsX, devMinReal,
                                        devMaxReal, devMinImag, devMaxImag,
                                        devPixelsY, devIterations);

    hipMemcpy((void*)pixelValues, (void*)devPixelValues,
               windowSize.x * windowSize.y * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    drawMandelbrot(renderer, windowSize, pixelValues);

    updateBoundaries();

    Sleep(200);
  }

  getchar();
  return 0;
}
