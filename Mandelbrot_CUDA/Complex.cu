
#include <hip/hip_runtime.h>

template <class T>
class Complex {
 private:
  T _real;
  T _imag;

 public:
  __device__ Complex() {
    this->_real = 0;
    this->_imag = 0;
  }

  __device__ Complex(T real, T imag) {
    this->_real = real;
    this->_imag = imag;
  }

  __device__ T real() { return this->_real; }

  __device__ T imag() { return this->_imag; }

  __device__ void real(T& real) { this->_real = real; }

  __device__ void imag(T& imag) { this->_imag = imag; }

  __device__ Complex<T> operator+(Complex<T>& c) {
    return Complex<T>(this->real() + c.real(), this->imag() + c.imag());
  }

  __device__ Complex<T> operator+(T& r) {
    return Complex<T>(this->real() + r, this->imag());
  }

  __device__ Complex<T> operator-(Complex<T>& c) {
    return Complex<T>(this->real() - c.real(), this->real() - c.real());
  }

  __device__ Complex<T> operator-(T& r) {
    return Complex<T>(this->real() - r, this->imag());
  }

  __device__ Complex<T> operator*(Complex<T>& c) {
    return Complex<T>(this->real() * c.real() - this->imag() * c.imag(),
                      this->real() * c.imag() + this->imag() * c.real());
  }

  __device__ Complex<T> operator*(T& r) {
    return Complex<T>(r * this->real(), r * this->imag());
  }

  __device__ Complex<T> operator/(Complex<T>& c) {
    T normalizer = c.real() * c.real() + c.imag() * c.imag();
    return Complex<T>(
        (this->real() * c.real() + this->imag() * c.imag()) / normalizer,
        (this->real() * c.imag() + this->imag() * c.real()) / normalizer);
  }

  __device__ Complex<T> operator/(T& r) {
    return Complex<T>(this->real() / r, this->imag() / r);
  }

  __device__ T abs() {
    return sqrt(this->real() * this->real() + this->imag() * this->imag());
  }
};
